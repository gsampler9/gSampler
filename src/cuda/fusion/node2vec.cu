#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <hiprand/hiprand_kernel.h>
#include "../cuda_common.h"
#include "../utils.h"
#include "node2vec.h"

#define MAX(x, y) ((x > y) ? x : y)
#define TID (threadIdx.x + blockIdx.x * blockDim.x)
#define LTID (threadIdx.x)
#define LID (threadIdx.x % 32)
#define WID (threadIdx.x / 32)
#define BLOCK_SIZE 256

using namespace cooperative_groups;

namespace gs {
namespace impl {
namespace fusion {

template <typename NType, typename EType>
__device__ __inline__ bool BinarySearch(NType* ptr, EType degree,
                                        NType target) {
  EType tmp_degree = degree;
  NType* tmp_ptr = ptr;

  int itr = 0;
  while (itr < 50) {
    if (tmp_ptr[tmp_degree / 2] == target) {
      return true;
    } else if (tmp_ptr[tmp_degree / 2] < target) {
      tmp_ptr += tmp_degree / 2;
      if (tmp_degree == 1) {
        return false;
      }
      tmp_degree = tmp_degree - tmp_degree / 2;
    } else {
      tmp_degree = tmp_degree / 2;
    }
    if (tmp_degree == 0) {
      return false;
    }
    itr++;
  }
  return false;
}

template <typename NType, typename EType>
__device__ __inline__ bool CheckConnect(NType* graph_indice,
                                        EType* graph_indptr, NType src,
                                        NType dst) {
  EType degree = graph_indptr[src + 1] - graph_indptr[src];
  /*
  IdType item = hipcub::UpperBound(graph_indice + graph_indptr[src], degree, dst);
  if (item == degree) {
    return false;
  } else {
    return true;
  }
  */

  if (BinarySearch(graph_indice + graph_indptr[src], degree, dst)) {
    return true;
  }
  return false;
}

template <typename EType, typename NType>
__global__ void _Node2VecKernel(const NType* seed_data, const int64_t num_seeds,
                                const uint64_t max_num_steps,
                                NType* graph_indice, EType* graph_indptr,
                                NType* out_traces_data, double p, double q) {
  // init
  hiprandState rng;
  uint64_t rand_seed = 7777777;
  hiprand_init(rand_seed + TID, 0, 0, &rng);
  double max_scale = MAX(p, q);

  NType total_num_threads = blockDim.x * gridDim.x;
  for (NType idx = TID; idx < num_seeds; idx += total_num_threads) {
    NType curr = seed_data[idx];
    out_traces_data[0 * num_seeds + idx] = curr;
  }

  // begin node2vec
  for (NType idx = TID; idx < num_seeds; idx += total_num_threads) {
    coalesced_group warp = coalesced_threads();

    NType curr = out_traces_data[0 * num_seeds + idx];
    NType lastV = TID;
    for (int step_idx = 0; step_idx < max_num_steps; step_idx++) {
      coalesced_group active = coalesced_threads();

      NType pick = -1;
      if (curr != -1) {
        const EType in_row_start = graph_indptr[curr];
        const EType deg = graph_indptr[curr + 1] - graph_indptr[curr];

        if (deg == 0) {
          pick = -1;
        } else if (deg > 1) {
          NType outV;
          do {
            int y = (int)floor(hiprand_uniform(&rng) * max_scale);

            bool early_reject = (y >= MAX(max_scale, 1.0));
            if (early_reject) continue;

            int x = (int)floor(hiprand_uniform(&rng) * deg);
            bool early_accept = (y < MIN(MIN(p, q), 1.0));
            if (early_accept) {
              outV = graph_indice[in_row_start + x];
              break;
            }

            double h;
            outV = graph_indice[in_row_start + x];
            if (lastV == outV) {
              h = p;
            } else if (CheckConnect(graph_indice, graph_indptr, lastV, outV)) {
              h = q;
            } else {
              h = 1.0;
            }
            if (y < h) break;
          } while (true);
          pick = outV;
        } else {
          // deg == 1
          pick = graph_indice[in_row_start];
        }
      }
      lastV = curr;
      curr = pick;
      out_traces_data[(step_idx + 1) * num_seeds + idx] = pick;
    }
  }
}

torch::Tensor FusedNode2VecCUDA(torch::Tensor seeds, int64_t walk_length,
                                torch::Tensor indices, torch::Tensor indptr,
                                double p, double q) {
  ID_TYPE_SWITCH(indptr.scalar_type(), EType, {
    ID_TYPE_SWITCH(indices.scalar_type(), NType, {
      const NType* seed_data = seeds.data_ptr<NType>();
      const int64_t num_seeds = seeds.numel();
      const uint64_t max_num_steps = (uint64_t)walk_length;
      int64_t outsize = num_seeds * (max_num_steps + 1);
      torch::Tensor out_traces_tensor =
          torch::full(outsize, -1, indices.options().device(torch::kCUDA));

      NType* out_traces_data = out_traces_tensor.data_ptr<NType>();
      dim3 block(BLOCK_SIZE);
      dim3 grid(num_seeds / BLOCK_SIZE + 1);
      _Node2VecKernel<<<grid, block>>>(seeds.data_ptr<NType>(), num_seeds,
                                       max_num_steps, indices.data_ptr<NType>(),
                                       indptr.data_ptr<EType>(),
                                       out_traces_data, p, q);
      return out_traces_tensor.reshape({seeds.numel(), -1});
    });
  });
  return torch::Tensor();
}

}  // namespace fusion
}  // namespace impl
}  // namespace gs